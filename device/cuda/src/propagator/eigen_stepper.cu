#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <cuda/propagator/eigen_stepper.cuh>
#include <propagator/propagator_options.hpp>
#include <cuda/utils/definitions.hpp>

namespace traccc {
namespace cuda {

using state = traccc::eigen_stepper::state;
__global__ void stepper_kernel(traccc::collection_view<state> stepper_state_view);

// Reserved to Xiangyang

bool traccc::cuda::eigen_stepper::rk4(host_collection<state>& states) {
    auto stepper_state_view = get_data(states);

    unsigned int num_threads = WARP_SIZE * 2;
    unsigned int num_blocks = stepper_state_view.items.size() / num_threads + 1;

    stepper_kernel<<<num_blocks, num_threads>>>(stepper_state_view);

    // cuda error check
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    return true;
}

__global__ void stepper_kernel(traccc::collection_view<state> stepper_state_view)
{
    traccc::device_collection<state> stepper_states_device(
        {stepper_state_view.items});

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    if (gid >= stepper_states_device.items.size()) {
        return;
    }

    traccc::eigen_stepper::rk4(stepper_states_device.items.at(gid));  
}


template void traccc::cuda::eigen_stepper::cov_transport<
    propagator_options<void_actor, void_aborter> >(
    host_collection<state>& state,
    host_collection<propagator_options<void_actor, void_aborter> >& options);

// Reserved to Johannes
template <typename propagator_options_t>
void traccc::cuda::eigen_stepper::cov_transport(
    host_collection<state>& state,
    host_collection<propagator_options_t>& options) {}

}  // namespace cuda
}  // namespace traccc
