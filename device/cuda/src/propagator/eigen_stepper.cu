#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <cuda/propagator/eigen_stepper.cuh>
#include <propagator/propagator_options.hpp>
#include <cuda/utils/definitions.hpp>

namespace traccc {
namespace cuda {

// Reserved to Xiangyang
bool traccc::cuda::eigen_stepper::rk4(host_collection<state>& state) {
    return true;
}

template void traccc::cuda::eigen_stepper::cov_transport<
    propagator_options<void_actor, void_aborter> >(
    host_collection<state>& state,
    host_collection<propagator_options<void_actor, void_aborter> >& options);

// Reserved to Johannes
template <typename propagator_options_t>
void traccc::cuda::eigen_stepper::cov_transport(
    host_collection<state>& state,
    host_collection<propagator_options_t>& options) {

    vecmem::cuda::managed_memory_resource mng_mr;
    auto state_view = get_data(state, &mng_mr);
    auto options_view = get_data(options, &mng_mr);

    unsigned int num_threads = WARP_SIZE * 2;
    unsigned int num_blocks = state_view.items.size() / num_threads + 1;

    // run the kernel
    cov_transport_kernel<propagator_options_t><<<num_blocks, num_threads>>>(state_view, options_view);

    // cuda error check
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

template <typename propagator_options_t>
__global__ void traccc::cuda::cov_transport_kernel(
    collection_view<state> states_view, collection_view<propagator_options_t> options_view) {
    
    //access collection
    device_collection<state> states_device({states_view.items});
    device_collection<propagator_options_t> options_device({options_view.items});

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    if (gid >= states_device.items.size()) {
        return;
    }

    //run cov_transport function
    traccc::eigen_stepper::cov_transport(
        states_device.items.at(gid), options_device.items.at(gid).mass);
}

}  // namespace cuda
}  // namespace traccc

