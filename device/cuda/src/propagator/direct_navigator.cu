#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <cuda/propagator/propagator.cuh>
#include <cuda/propagator/direct_navigator.cuh>
#include <propagator/eigen_stepper.hpp>
#include <propagator/direct_navigator.hpp>
#include <cuda/utils/definitions.hpp>
#include <propagator/propagator_options.hpp>

namespace traccc {
namespace cuda {


// kernel declaration
template <typename propagator_state_t, typename surface_t>
__global__ void status_kernel(collection_view<propagator_state_t> states_view,
			      collection_view<surface_t> surfaces_view);    

    
// explicit type instantiation    
using truth_propagator = typename traccc::cuda::propagator<traccc::eigen_stepper, traccc::direct_navigator>;
using void_propagator_options = typename traccc::propagator_options<void_actor, void_aborter>;
using void_multi_state = typename truth_propagator::multi_state< void_propagator_options >;    

template void traccc::cuda::direct_navigator::status<void_multi_state, surface>(void_multi_state& state, host_collection< surface >& surfaces);


template <typename propagator_state_t, typename surface_t>
void traccc::cuda::direct_navigator::status(propagator_state_t& state,
					    host_collection<surface_t>& surfaces){

    auto states_view = get_data(state.states);
    auto surfaces_view = get_data(surfaces);

    unsigned int num_threads = WARP_SIZE * 2;
    unsigned int num_blocks = states_view.items.size() / num_threads + 1;
    
    // run the kernel
    status_kernel< typename propagator_state_t::state_t, surface_t > 
        <<<num_blocks, num_threads>>>(states_view, surfaces_view);

    // cuda error check
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    
}

template <typename propagator_state_t, typename surface_t>
__global__ void status_kernel(collection_view<propagator_state_t> states_view,
			      collection_view<surface_t> surfaces_view){

    device_collection<propagator_state_t> states_device({states_view.items});
    device_collection<surface_t> surfaces_device({surfaces_view.items});
    
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    if (gid >= states_device.items.size()) {
        return;
    }

    traccc::direct_navigator::status(states_device.items.at(gid),
                                     &surfaces_device.items.at(0));
    
}
    
}  // namespace cuda
}  // namespace traccc
