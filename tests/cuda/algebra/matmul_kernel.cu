#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <hip/hip_runtime.h>
#include "matmul_kernel.cuh"

__global__ void matmul_kernel(
    vecmem::data::vector_view< Acts::BoundSymMatrix > A_view,
    vecmem::data::vector_view< Acts::BoundSymMatrix > B_view,
    vecmem::data::vector_view< Acts::BoundSymMatrix > C_view);

__global__ void matmul_kernel(
    vecmem::data::vector_view< Acts::FreeSymMatrix > A_view,
    vecmem::data::vector_view< Acts::FreeSymMatrix > B_view,
    vecmem::data::vector_view< Acts::FreeSymMatrix > C_view);

void matmul(int n_matrix,
	    vecmem::data::vector_view< Acts::BoundSymMatrix > A_view,
	    vecmem::data::vector_view< Acts::BoundSymMatrix > B_view,
	    vecmem::data::vector_view< Acts::BoundSymMatrix > C_view,
	    float& time){

    unsigned int num_threads = WARP_SIZE*2;
    unsigned int num_blocks = n_matrix/num_threads + 1;

    //--TIME--------------------------------------
    float elapsed=0;
    hipEvent_t start, stop;    
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&stop));
    CUDA_ERROR_CHECK(hipEventRecord(start, 0));
    //--------------------------------------------
    
    matmul_kernel<<< num_blocks, num_threads >>>(A_view, B_view, C_view);

    // cuda error check    
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    
    //--TIME--------------------------------------
    CUDA_ERROR_CHECK(hipEventRecord(stop, 0));
    CUDA_ERROR_CHECK(hipEventSynchronize (stop) );    
    CUDA_ERROR_CHECK(hipEventElapsedTime(&elapsed, start, stop) );    
    CUDA_ERROR_CHECK(hipEventDestroy(start));
    CUDA_ERROR_CHECK(hipEventDestroy(stop));
    elapsed*=0.001; // ms->sec
    time += elapsed;
    //--------------------------------------------
    
}

void matmul(int n_matrix,
	    vecmem::data::vector_view< Acts::FreeSymMatrix > A_view,
	    vecmem::data::vector_view< Acts::FreeSymMatrix > B_view,
	    vecmem::data::vector_view< Acts::FreeSymMatrix > C_view,
	    float& time){

    unsigned int num_threads = WARP_SIZE*2;
    unsigned int num_blocks = n_matrix/num_threads + 1;

    //--TIME--------------------------------------
    float elapsed=0;
    hipEvent_t start, stop;    
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&stop));
    CUDA_ERROR_CHECK(hipEventRecord(start, 0));
    //--------------------------------------------
    
    matmul_kernel<<< num_blocks, num_threads >>>(A_view, B_view, C_view);

    //--TIME--------------------------------------
    CUDA_ERROR_CHECK(hipEventRecord(stop, 0));
    CUDA_ERROR_CHECK(hipEventSynchronize (stop) );    
    CUDA_ERROR_CHECK(hipEventElapsedTime(&elapsed, start, stop) );    
    CUDA_ERROR_CHECK(hipEventDestroy(start));
    CUDA_ERROR_CHECK(hipEventDestroy(stop));
    elapsed*=0.001; // ms->sec
    time += elapsed;
    //--------------------------------------------
    
    // cuda error check    
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
}


__global__ void matmul_kernel(
    vecmem::data::vector_view< Acts::BoundSymMatrix > A_view,
    vecmem::data::vector_view< Acts::BoundSymMatrix > B_view,
    vecmem::data::vector_view< Acts::BoundSymMatrix > C_view){
    
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    vecmem::device_vector< Acts::BoundSymMatrix > A_dev(A_view);
    vecmem::device_vector< Acts::BoundSymMatrix > B_dev(B_view);
    vecmem::device_vector< Acts::BoundSymMatrix > C_dev(C_view);

    if (gid >= A_dev.size()){
	return;
    }
        
    C_dev.at(gid) = A_dev.at(gid)*B_dev.at(gid);
}

__global__ void matmul_kernel(
    vecmem::data::vector_view< Acts::FreeSymMatrix > A_view,
    vecmem::data::vector_view< Acts::FreeSymMatrix > B_view,
    vecmem::data::vector_view< Acts::FreeSymMatrix > C_view){

    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    vecmem::device_vector< Acts::FreeSymMatrix > A_dev(A_view);
    vecmem::device_vector< Acts::FreeSymMatrix > B_dev(B_view);
    vecmem::device_vector< Acts::FreeSymMatrix > C_dev(C_view);

    if (gid >= A_dev.size()){
	return;
    }
        
    C_dev.at(gid) = A_dev.at(gid)*B_dev.at(gid);
}
