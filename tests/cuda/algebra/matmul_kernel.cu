#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <hip/hip_runtime.h>
#include "matmul_kernel.cuh"

// kernel declaration
template < typename matrix_t >
__global__ void matmul_kernel(
    vecmem::data::vector_view< matrix_t > A_view,
    vecmem::data::vector_view< matrix_t > B_view,
    vecmem::data::vector_view< matrix_t > C_view);

// explicit instantiation of matmul function
template 
void matmul<Acts::BoundSymMatrix>(int n_matrix,
	    vecmem::data::vector_view< Acts::BoundSymMatrix > A_view,
	    vecmem::data::vector_view< Acts::BoundSymMatrix > B_view,
	    vecmem::data::vector_view< Acts::BoundSymMatrix > C_view,
	    float& elapsed);

// explicit instantiation of matmul function
template 
void matmul<Acts::FreeSymMatrix>(int n_matrix,
	    vecmem::data::vector_view< Acts::FreeSymMatrix > A_view,
	    vecmem::data::vector_view< Acts::FreeSymMatrix > B_view,
	    vecmem::data::vector_view< Acts::FreeSymMatrix > C_view,
	    float& elapsed);

// Definition of matmul function
template < typename matrix_t >
void matmul(int n_matrix,
	    vecmem::data::vector_view< matrix_t > A_view,
	    vecmem::data::vector_view< matrix_t > B_view,
	    vecmem::data::vector_view< matrix_t > C_view,
	    float& time){

    unsigned int num_threads = WARP_SIZE*2;
    unsigned int num_blocks = n_matrix/num_threads + 1;

    //--TIME--------------------------------------
    float elapsed=0;
    hipEvent_t start, stop;    
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&stop));
    CUDA_ERROR_CHECK(hipEventRecord(start, 0));
    //--------------------------------------------
    
    matmul_kernel<<< num_blocks, num_threads >>>(A_view, B_view, C_view);

    // cuda error check    
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    
    //--TIME--------------------------------------
    CUDA_ERROR_CHECK(hipEventRecord(stop, 0));
    CUDA_ERROR_CHECK(hipEventSynchronize (stop) );    
    CUDA_ERROR_CHECK(hipEventElapsedTime(&elapsed, start, stop) );    
    CUDA_ERROR_CHECK(hipEventDestroy(start));
    CUDA_ERROR_CHECK(hipEventDestroy(stop));
    elapsed*=0.001; // ms->sec
    time += elapsed;
    //--------------------------------------------
    
}

// kernel definition
template < typename matrix_t >
__global__ void matmul_kernel(
    vecmem::data::vector_view< matrix_t > A_view,
    vecmem::data::vector_view< matrix_t > B_view,
    vecmem::data::vector_view< matrix_t > C_view){

    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    vecmem::device_vector< matrix_t > A_dev(A_view);
    vecmem::device_vector< matrix_t > B_dev(B_view);
    vecmem::device_vector< matrix_t > C_dev(C_view);

    if (gid >= A_dev.size()){
	return;
    }
        
    C_dev.at(gid) = A_dev.at(gid)*B_dev.at(gid);
    
}
